#include "hip/hip_runtime.h"
#include <math.h>
#include <fstream>

#include "helper.h"

using namespace std;

// geoDistance computes geographical distance (lat1, lat1) and (lat2, lon2)
__device__ float geoDistance(float lat1, float lon1, float lat2, float lon2)
{
    float phi1 = (90.f - lat1) * DEGREE_TO_RADIANS;
    float phi2 = (90.f - lat2) * DEGREE_TO_RADIANS;

    float theta1 = lon1 * DEGREE_TO_RADIANS;
    float theta2 = lon2 * DEGREE_TO_RADIANS;

    float cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) + cos(phi1) * cos(phi2);
    if (cs > 1) {
        cs = 1;
    } else if (cs < -1) {
        cs = -1;
    }

    return 6371.f * acos(cs);
}

#include "hip/hip_runtime.h"
#include <stdlib.h>

#include <fstream>
#include <iostream>
#include <vector>

#include "helper.h"

using namespace std;

#define PREALLOC_SIZE 1000
#define BLOCK_SIZE_GPU 256

/**
 * @brief Kernel function where each GPU thread gets assigned a city and finds 
 * all of the other cities in the range specified, hence adding their 
 * populations.
 *
 * @param total_pops array of cities' total population count
 * @param lats array of cities' latitudes
 * @param lons array of cities' longitudes
 * @param pops array of cities' population count
 * @param kmRange radius of search
 * @param n number of cities
 * @return __global__ none
 */
__global__ void add_populations(int *total_pops, const float *lats,
                                const float *lons, const int *pops,
                                const int kmRange, const size_t n) {
    // Get current city assigned to the GPU thread
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int *src_total_pop = &total_pops[idx];
    const float *src_lat = &lats[idx];
    const float *src_lon = &lons[idx];
    const int *src_pop = &pops[idx];

    // Check for outer bound
    if (idx >= n) return;

    /*
    * Finds the cities in range of the thread's city and adds their population.
    * The city that is found will have the thread's population added to it, 
    * thus the check is done only with further cities in the vector.
    */
    for (unsigned int i = idx + 1; i < n; ++i) {
        if (geoDistance(*src_lat, *src_lon, lats[i], lons[i]) <= kmRange) {
            atomicAdd(src_total_pop, pops[i]);
            atomicAdd(&total_pops[i], *src_pop);
        }
    }
}

/**
 * @brief CPU function that processes cities' data and outputs total population
 * count of each city's radius.
 *
 * @param kmRange city radius
 * @param fileIn input file name to read cities' data
 * @param fileOut output file name to write cities' total population count
 */
void process_cities(float kmRange, const char *fileIn, const char *fileOut) {
    string geon;
    float lat, lon;
    int pop;
    float *latsHost, *latsDevice, *lonsHost, *lonsDevice;
    int *popsHost, *popsDevice, *total_popsHost, *total_popsDevice;
    unsigned int num_cities = 0, capacity = PREALLOC_SIZE;
    vector<string> geons;

    // Open IO files
    ifstream ifs(fileIn);
    ofstream ofs(fileOut);

    // Initialize host heap memory
    latsHost = (float *)malloc(capacity * sizeof(float));
    lonsHost = (float *)malloc(capacity * sizeof(float));
    popsHost = (int *)malloc(capacity * sizeof(int));
    total_popsHost = (int *)malloc(capacity * sizeof(int));

    if (total_popsHost == NULL || popsHost == NULL || lonsHost == NULL ||
        latsHost == NULL) {
        cout << "Host memory allocation error.\n";
        return;
    }

    // Read data from input file
    while (ifs >> geon >> lat >> lon >> pop) {
        // Check if more memory needs to be reserved
        if (num_cities >= capacity) {
            capacity *= 2;
            latsHost = (float *)realloc(latsHost, sizeof(float) * capacity);
            lonsHost = (float *)realloc(lonsHost, sizeof(float) * capacity);
            popsHost = (int *)realloc(popsHost, sizeof(int) * capacity);
            total_popsHost =
                (int *)realloc(total_popsHost, sizeof(int) * capacity);

            if (total_popsHost == NULL || popsHost == NULL ||
                lonsHost == NULL || latsHost == NULL) {
                cout << "Host memory reallocation error.\n";
                return;
            }
        }

        // Add data to vectors
        geons.push_back(geon);
        latsHost[num_cities] = lat;
        lonsHost[num_cities] = lon;
        popsHost[num_cities] = pop;

        // Add city's population to total population count
        total_popsHost[num_cities] = pop;

        ++num_cities;
    }

    // Initialize device heap memory
    hipMalloc((void **)&latsDevice, num_cities * sizeof(float));
    hipMalloc((void **)&lonsDevice, num_cities * sizeof(float));
    hipMalloc((void **)&popsDevice, num_cities * sizeof(int));
    hipMalloc((void **)&total_popsDevice, num_cities * sizeof(int));

    if (total_popsDevice == NULL || popsDevice == NULL || lonsDevice == NULL ||
        latsDevice == NULL) {
        cout << "Device memory allocation error.\n";
        return;
    }

    // Copy data from Host to Device
    hipMemcpy(latsDevice, latsHost, num_cities * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(lonsDevice, lonsHost, num_cities * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(popsDevice, popsHost, num_cities * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(total_popsDevice, total_popsHost, num_cities * sizeof(int),
               hipMemcpyHostToDevice);

    // Choose block and grid size
    const size_t block_size = BLOCK_SIZE_GPU;
    size_t blocks_no = (num_cities + block_size - 1) / block_size;

    // Launch GPU Kernel to compute total population counts
    add_populations<<<blocks_no, block_size>>>(total_popsDevice, latsDevice,
                                               lonsDevice, popsDevice, kmRange,
                                               num_cities);
    if (hipSuccess != hipDeviceSynchronize()) {
        cout << "Cuda Synchronize\n";
        return;
    }

    // Copy results from the Device to the Host
    hipMemcpy(total_popsHost, total_popsDevice, num_cities * sizeof(int),
               hipMemcpyDeviceToHost);

    // Print results to output file
    for (unsigned int i = 0; i < num_cities; ++i) {
        ofs << total_popsHost[i] << '\n';
    }

    // Free Host memory
    free(latsHost);
    free(lonsHost);
    free(popsHost);
    free(total_popsHost);

    // Free Device Memory
    hipFree(latsDevice);
    hipFree(lonsDevice);
    hipFree(popsDevice);
    hipFree(total_popsDevice);

    // Close IO files
    ifs.close();
    ofs.close();
}

int main(int argc, char **argv) {
    DIE(argc == 1, "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE((argc - 1) % 3 != 0, "./accpop <kmrange1> <file1in> <file1out> ...");

    for (int argcID = 1; argcID < argc - 3; argcID += 3) {
        float kmRange = atof(argv[argcID]);
        process_cities(kmRange, argv[argcID + 1], argv[argcID + 2]);
    }
}
